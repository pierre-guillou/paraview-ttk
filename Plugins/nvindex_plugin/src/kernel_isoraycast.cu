#include "hip/hip_runtime.h"
/* Copyright 2018 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// # RTC Kernel:
// ** Volume Alpha-Gradient Shading Raycaster **

// # Summary:
// Computes the alpha gradient of the volume & transfer function along a ray and applies Phong
// shading above a user defined threshold.

// Define the user-defined data structure
struct Isoraycast_params
{
  // common lighting params
  int light_mode;  // 0=headlight, 1=orbital
  float angle;     // 0.0 angle
  float elevation; // 0.0 elevation

  float diff_h; // 0.0f [0, 2] [GUI] transfer function gradient threshold to trigger phong shading
  uint use_shading; // 1 [GUI] use local phong-blinn model

  // shading [GUI / scene]
  float3 spec_color; // make_float3(1.0f) specular color
  float spec_fac;    // specular factor
  float shininess;   // 50.0f shininess parameter (phong)
  float amb_fac;     // 0.2f ambient factor
  float diff_exp;    // 2.0f diffuse falloff (like edge enhance)
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
  NV_IDX_VOLUME_SAMPLE_PROGRAM

public:
  const Isoraycast_params* m_isoraycast_params; // define variables to bind user-defined buffer to

public:
  NV_IDX_DEVICE_INLINE_MEMBER
  void initialize()
  {
    // Bind the contents of the buffer slot 0 to the variable
    m_isoraycast_params = state.bind_parameter_buffer<Isoraycast_params>(0);
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  int execute(const Sample_info_self&   sample_info,
                    Sample_output&      sample_output)
  {
    const auto& volume = state.self;
    const float3& sample_position = sample_info.sample_position;
    const float3 ray_dir = sample_info.ray_direction;
    const Colormap& colormap = volume.get_colormap();

    const float volume_sample = volume.sample<float>(sample_position);

    // get spatial sample points for each dimensions
    const float rh = state.self.get_stepsize_min(); // ray sampling difference
    const float vs_dr_p = volume.sample<float>(sample_position + ray_dir * rh);
    const float vs_dr_n = volume.sample<float>(sample_position - ray_dir * rh);

    const float4 c0 = colormap.lookup(vs_dr_p);
    const float4 c1 = colormap.lookup(volume_sample);
    const float4 c2 = colormap.lookup(vs_dr_n);

    // sum up threshold exceeds in both directions
    const float smin = min(c0.w, min(c1.w, c2.w));
    const float smax = max(c0.w, max(c1.w, c2.w));

    // check for iso intersections
    if ((smax - smin) >= m_isoraycast_params->diff_h)
    {
      // valid intersection found
      if (m_isoraycast_params->use_shading)
      {
        sample_output.color = blinn_shader(sample_position, c1, ray_dir);
      }
      else
      {
        // use sample color
        sample_output.color = c1;
        sample_output.color.w = 1.0f;
      }

      return NV_IDX_PROG_OK;
    }
    else
    {
      // use sample color
      sample_output.color = c1;
    }

    return NV_IDX_PROG_OK;
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  void gamma(float4& out_color, const float gamma)
  {
    // apply gamma correction
    out_color.x = powf(out_color.x, float(1.0f / gamma));
    out_color.y = powf(out_color.y, float(1.0f / gamma));
    out_color.z = powf(out_color.z, float(1.0f / gamma));
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  float4 blinn_shader(const float3& sample_position, const float4& sample_color, const float3& ray_dir)
  {
    // get isosurface normal
    const float3 iso_normal = -normalize(state.self.get_gradient(sample_position));

    float3 light_dir;
    if (m_isoraycast_params->light_mode == 0)
    {
      light_dir = ray_dir;
    }
    else
    {
      const float theta = m_isoraycast_params->angle;
      const float phi = m_isoraycast_params->elevation;
      light_dir = make_float3(sinf(phi) * cosf(theta), sinf(phi) * sinf(theta), cosf(phi));
    }

    // diffuse term correction (edge enhance)
    const float diff_amnt = fabsf(dot(light_dir, iso_normal));
    const float diff_fac = powf(diff_amnt, m_isoraycast_params->diff_exp);

    // specular term
    float spec_amnt = 0.0f;

    if (diff_amnt > 0.0f)
    {
      const float3 H = normalize(light_dir + ray_dir);
      const float NH = fabsf(dot(H, iso_normal));
      spec_amnt = powf(NH, m_isoraycast_params->shininess);
    }

    // compute final color (RGB)
    const float3 shade_color =
      make_float3(sample_color) * (m_isoraycast_params->amb_fac + diff_amnt * diff_fac) +
      m_isoraycast_params->spec_color * (spec_amnt * m_isoraycast_params->spec_fac);

    return clamp(
      make_float4(shade_color.x, shade_color.y, shade_color.z, sample_color.w), 0.0f, 1.0f);
  }
}; // class Volume_sample_program
