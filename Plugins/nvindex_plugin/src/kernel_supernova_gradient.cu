#include "hip/hip_runtime.h"
/* Copyright 2018 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// # RTC Kernel:
// ** Volume Gradient Colormapping **
// # Summary: Compute the volume gradient and map to the given colormap.

struct Supernova_gradient_params
{
    float gradient_scale;       // gradient value (0.5) [0,1]
    float screen_gamma;         // (0.55)
    int   color_method;         // Gradient Color method (3):
                                // 0: use z gradient only
                                // 1: use x,y gradient only
                                // 2: use gradient magnitude
                                // 3: darken sample color by magnitude
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
    NV_IDX_VOLUME_SAMPLE_PROGRAM

    const float min_alpha       = 0.001f;    // min alpha to trigger gradient computation
    const float dh              = 2.0f;     // finite-differencing stepsize

   // highlight core region (fixed to dataset)
    const float4 nova_color     = make_float4(1.0f, 0.4f , 0.0f, 1.0f);
    const float3 nova_center    = make_float3(316.5f);
    const float nova_max_dist   = 8.0f;
    const float nova_falloff_exp = 1.75f;

    // Use gradient to set transparency
    const bool use_grad_alpha   = 1;

    const Colormap colormap = state.self.get_colormap();
    const Supernova_gradient_params*  m_params;

public:
    NV_IDX_DEVICE_INLINE_MEMBER
    void initialize()
    {
        m_params = state.bind_parameter_buffer<Supernova_gradient_params>(0);
    }

    NV_IDX_DEVICE_INLINE_MEMBER
    int execute(
        const Sample_info_self&  sample_info,
              Sample_output&     sample_output)
    {
        const float3& sample_position = sample_info.sample_position;
        const float3& scene_position = sample_info.scene_position;

        // highlight core region
        const float dist = length(nova_center - scene_position);
        if (dist < nova_max_dist)
        {
            float nd = 1.0f - (dist / nova_max_dist);
            nd = powf(nd, nova_falloff_exp);

            sample_output.color = (1.0f - nd) * make_float4(1.0f) + nd * nova_color;
            sample_output.color.w = nd;

            return NV_IDX_PROG_OK;
        }

        // sample volume, local gradient and color
        const float  volume_sample = state.self.sample<float>(sample_position);
        const float4 sample_color  = colormap.lookup(volume_sample);

        // check if sample can be skipped
        if (sample_color.w < min_alpha) return NV_IDX_PROG_DISCARD_SAMPLE;

        // compute volume gradient
        const float3 vol_grad = state.self.get_gradient(sample_position, dh);

        // scale gradient by user input
        float grad_scale = m_params[0].gradient_scale;

        if (m_params[0].color_method == 0)
        {
            // color by height gradient
            sample_output.color  = colormap.lookup(vol_grad.z * grad_scale);
        }
        else if (m_params[0].color_method == 1)
        {
            // color by x,y gradient
            const float vs_xy_mag = sqrt(pow(vol_grad.x,2.0f) * pow(vol_grad.y,2.0f));
            sample_output.color  = colormap.lookup(vs_xy_mag * grad_scale);
        }
        else if (m_params[0].color_method == 2)
        {
            // color by gradient magnitude
            const float grad_mag = length(vol_grad);
            sample_output.color  = colormap.lookup(grad_mag * grad_scale);
        }
        else
        {
            // shade color by gradient magnitude
            const float grad_mag = length(vol_grad);
            sample_output.color  = clamp(sample_color * (grad_mag * grad_scale),0.0f,1.0f);
        }

        if (!use_grad_alpha) sample_output.color.w = sample_color.w;

        // apply gamma correction
        sample_output.color = gamma_correct(sample_output.color, m_params[0].screen_gamma);

        return NV_IDX_PROG_OK;
    }
}; // class Volume_sample_program
