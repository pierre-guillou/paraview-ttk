#include "hip/hip_runtime.h"
/* Copyright 2018 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// # RTC Kernel:
// ** Volume Single-Scattering **

// # Summary:
// Trace an additional shadow ray in a user-controlled direction to darken the current sample
// color.
// NOTE: creates artifacts if shadow ray crosses subregion boundaries

// Define the user-defined data structure
struct Single_scattering_params
{
  // common lighting params
  int light_mode;       // 0=headlight, 1=orbital
  float angle;          // 0.0f [GUI] shadow ray angle around axis
  float elevation;      // pi/2 [GUI] shadow ray elevation around axis
  int steps;            // 50 [GUI] shadow ray samples (maximum)
  float light_distance; // 100.0f [GUI] light traveling distance (maximum)
  float min_alpha;      // 0.1f [GUI] alpha threshold for scattering
  float max_shadow;     // 0.2f [GUI] darkest shadow factor
  float shadow_exp;     // 1.5f [GUI] shadow dampening
  float shadow_offset;  // 0.05f offset shadow ray (avoids voxel self darkening)

  // shading
  uint two_sided;   // true
  uint use_shading; // true [GUI] use local shading
  float amb_fac;    // 0.8f ambient factor
  float spec_fac;   // 0.5f specular factor
  float shininess;  // 50   speculat shininess

  // volume
  float dh;             // 1.0f finite difference (gradient approximation)
  uint volume_id;       // 0u volume to sample
  float min_samp_alpha; // 0.01f minimum processing alpha

  float3 dummy;
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
  NV_IDX_VOLUME_SAMPLE_PROGRAM

public:
  const Single_scattering_params*
    m_single_scattering_params; // define variables to bind user-defined buffer to

public:
  NV_IDX_DEVICE_INLINE_MEMBER
  void initialize()
  {
    // Bind the contents of the buffer slot 0 to the variable
    m_single_scattering_params = state.bind_parameter_buffer<Single_scattering_params>(0);
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  int execute(const Sample_info_self&   sample_info,
                    Sample_output&      sample_output)
  {
    // retrieve parameter buffer contents (fixed values in code definition)
    const auto& volume = state.scene.access_by_id<Regular_volume>(m_single_scattering_params->volume_id);
    const float3& sample_position = sample_info.sample_position;
    const Colormap& colormap = state.self.get_colormap();

    // sample volume and colormap
    const float volume_sample = volume.sample<float>(sample_position);
    const float4 sample_color = colormap.lookup(volume_sample);

    if (sample_color.w < m_single_scattering_params->min_samp_alpha)
    {
      sample_output.color = sample_color;
      return NV_IDX_PROG_DISCARD_SAMPLE;
    }

    // setup light direction
    float3 light_dir;
    if (m_single_scattering_params->light_mode == 0)
    {
      light_dir = state.scene.camera.get_to() * m_single_scattering_params->light_distance;
    }
    else
    {
      const float theta = m_single_scattering_params->angle;
      const float phi = m_single_scattering_params->elevation;
      light_dir = make_float3(sinf(phi) * cosf(theta), sinf(phi) * sinf(theta), cosf(phi)) *
        m_single_scattering_params->light_distance;
    }

    const float3 world_pos = sample_info.scene_position;
    const float3 light_pos = world_pos + light_dir;
    const float3 light_off = light_dir *
      m_single_scattering_params->shadow_offset; // offset to avoid self darkening of voxels

    // init output color
    if (m_single_scattering_params->use_shading)
    {
      sample_output.color = blinn_shading(
        sample_position, sample_color, -normalize(light_dir), sample_info.ray_direction, m_single_scattering_params->dh);
      sample_output.color.w = sample_color.w;
    }
    else
    {
      sample_output.color = sample_color;
    }

    // check for iso intersections
    if (sample_color.w > m_single_scattering_params->min_alpha)
    {
      // init shadow value
      float acc_shadow = 1.0f;
      int scc = 0;

      // iterate shadow samples
      for (int sc = 0; sc < m_single_scattering_params->steps; sc++)
      {
        scc++;
        float sample_value = 0.0f;

        const float st = (float)(sc) / (float)(m_single_scattering_params->steps - 1.0f);

        const float3 sray_pos = world_pos + st * light_dir + light_off;
        const float3 sub_pos = transform_point(state.self.get_scene_to_sample_transform(), sray_pos);

        sample_value = volume.sample<float>(sub_pos);

        // if (volume.is_inside(world_pos))
        if (volume.is_inside(sub_pos))
        {
          const float4 rs_color = colormap.lookup(sample_value);
          const float voxel_alpha = rs_color.w;

          // accumulate darkening along ray
          acc_shadow *= (1.0f - powf(voxel_alpha, m_single_scattering_params->shadow_exp));

          if (acc_shadow < m_single_scattering_params->max_shadow)
          {
            acc_shadow = m_single_scattering_params->max_shadow;
            break;
          }
        }
        else
          break;
      }

      // apply shadow correction
      sample_output.color *= fmaxf(fminf(acc_shadow, 1.0f), 0.0f);
      sample_output.color.w = sample_color.w;
    }

    return NV_IDX_PROG_OK;
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  float4 blinn_shading(const float3& sample_position,
    const float4& sample_color,
    const float3& light_dir,
    const float3& view_dir,
    const float dh)
  {
    // define parameters
    const float3 diff_color = make_float3(sample_color);
    const float3 spec_color = make_float3(1.0f); // specular color
    const float screen_gamma = 0.7f;             // gamma correction parameter

    // get R3 gradient vector
    const float3 vs_grad = state.self.get_gradient(sample_position, dh); // compute R3 gradient
    const float3 iso_normal = -normalize(vs_grad);                       // get isosurface normal

    const float diff_amnt = fabsf(dot(light_dir, iso_normal)); // two sided shading
    float spec_amnt = 0.0f;

    if (diff_amnt > 0.0f)
    {
      // this is blinn phong
      float3 H = normalize(light_dir + view_dir);
      float NH = fabsf(dot(H, iso_normal)); // two sided shading
      spec_amnt = powf(NH, m_single_scattering_params->shininess);
    }

    // compute final color (RGB)
    const float3 color_linear = diff_color * (m_single_scattering_params->amb_fac + diff_amnt) +
      spec_color * (spec_amnt * m_single_scattering_params->spec_fac);

    // apply gamma correction
    float4 color_gcorrect = make_float4(color_linear.x, color_linear.y, color_linear.z, 1.0f);
    color_gcorrect.x = powf(color_gcorrect.x, float(1.0f / screen_gamma));
    color_gcorrect.y = powf(color_gcorrect.y, float(1.0f / screen_gamma));
    color_gcorrect.z = powf(color_gcorrect.z, float(1.0f / screen_gamma));
    color_gcorrect.w = sample_color.w;

    return clamp(color_gcorrect);
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  float4 clamp(const float4& color)
  {
    return make_float4(max(min(color.x, 1.0f), 0.0f),
      max(min(color.y, 1.0f), 0.0f),
      max(min(color.z, 1.0f), 0.0f),
      max(min(color.w, 1.0f), 0.0f));
  }
}; // class Volume_sample_program
